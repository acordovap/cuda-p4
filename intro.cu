#include "hip/hip_runtime.h"
/*
 *
 * Programa de Introducción a los conceptos de CUDA
 * Mariana Hernández
 * Alan Córdova
 *
 *
 */

#include <stdio.h>
#include <stdlib.h>

/* Declaración de métodos/


/* Utilidad para checar errores de CUDA */
void checkCUDAError(const char*);

/* Kernel para sumar dos vectores en un sólo bloque de hilos */
__global__ void vect_add(int *d_a, int *d_b, int *d_c)
{
    /* Part 2B: Implementación del kernel para realizar la suma de los vectores en el GPU */
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // if (i < N)
        d_c[i] = d_a[i] + d_b[i];
}

/* Versión de múltiples bloques de la suma de vectores */
__global__ void vect_add_multiblock(int *d_a, int *d_b, int *d_c)
{
    /* Part 2C: Implementación del kernel pero esta vez permitiendo múltiples bloques de hilos. */
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // if (i < N)
        d_c[i] = d_a[i] + d_b[i];
}

/* Numero de elementos en el vector */
#define ARRAY_SIZE 256

/*
 * Número de bloques e hilos
 * Su producto siempre debe ser el tamaño del vector (arreglo).
 */
#define NUM_BLOCKS  1
#define THREADS_PER_BLOCK 256

/* Main routine */
int main(int argc, char *argv[])
{
    int *a, *b, *c, *d; /* Arreglos del CPU */
    int *d_a, *d_b, *d_c, *d_d;/* Arreglos del GPU */

    int i;
    size_t sz = ARRAY_SIZE * sizeof(int);

    /*
     * Reservar memoria en el cpu
     */
    a = (int *) malloc(sz);
    b = (int *) malloc(sz);
    c = (int *) malloc(sz);
    d = (int *) malloc(sz);

    /*
     * Parte 1A:Reservar memoria en el GPU
     */
    hipMalloc(&d_a, sz);
    hipMalloc(&d_b, sz);
    hipMalloc(&d_c, sz);
    hipMalloc(&d_d, sz);

    /* inicialización */
    for (i = 0; i < ARRAY_SIZE; i++) {
        a[i] = i;
        b[i] = ARRAY_SIZE - i;
        c[i] = 0;
        d[i] = 0;
    }

    /* Parte 1B: Copiar los vectores del CPU al GPU */
    hipMemcpy(d_a, a, sz, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sz, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, sz, hipMemcpyHostToDevice);
    hipMemcpy(d_d, d, sz, hipMemcpyHostToDevice);

    /* run the kernel on the GPU */
    /* Parte 2A: Configurar y llamar los kernels */
    /* dim3 dimGrid( ); */
    /* dim3 dimBlock( ); */
    /* vect_add<<< , >>>( ); */

    //invocamos kernel
    int threadsPerBlock = 64; // ARRAY_SIZE/NUM_BLOCKS
    int blocksPerGrid = 4; // nuevo NUM_BLOCKS

    // Para obtener tiempos de ejecucion del kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vect_add<<< NUM_BLOCKS , THREADS_PER_BLOCK >>> (d_a, d_b, d_c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Tiempo de vect_add: \t %f \n", milliseconds);


    hipEventRecord(start);
    vect_add_multiblock<<< blocksPerGrid , threadsPerBlock >>> (d_a, d_b, d_d);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Tiempo de vect_add_multiblock (4 bloques): \t %f \n", milliseconds);

    /* Esperar a que todos los threads acaben y checar por errores */
    hipDeviceSynchronize();
    checkCUDAError("kernel invocation");

    /* Part 1C: copiar el resultado de nuevo al CPU */
    hipMemcpy(a, d_a, sz, hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, sz, hipMemcpyDeviceToHost);
    hipMemcpy(c, d_c, sz, hipMemcpyDeviceToHost);
    hipMemcpy(d, d_d, sz, hipMemcpyDeviceToHost);

    checkCUDAError("memcpy");

    /* print out the result */
    printf("Results: ");
    for (i = 0; i < ARRAY_SIZE; i++) {
      printf("%d, ", c[i]);
    }

    for (i = 0; i < ARRAY_SIZE; i++) {
      printf("%d, ", d[i] );
    }
    printf("\n\n");

    /* Parte 1D: Liberar los arreglos */
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);

    free(a);
    free(b);
    free(c);
    free(d);

    return 0;
}


/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
