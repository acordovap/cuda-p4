
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 512
#define MAX_ERR 1

__global__ void vector_add(float *out, float *a, float *b, int n){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  //printf("index: %d\n", index);
  for(int i = index; i < n; i+=blockDim.x){
    printf("i=%d \n", i);
    out[i] = a[i] + b[i];
  }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
      a[i] = 1.0f; b[i] = 2.0f;
    }

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host t  o device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function

    vector_add<<</*1*/(N+256)/256,256>>>(d_out, d_a, d_b, N);

    // Transfer data from device to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Print results
    for(int i=0; i < N; i++){
      //if(fabs(out[i] - a[i] - b[i]) < MAX_ERR )
      //printf("failed");
      printf("%i.- %f = %f + %f \n", i, out[i], a[i], b[i]);
    }

    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);

}
