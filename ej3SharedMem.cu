/*
 *  Ejercicio 5 Práctica 4: CUDA
 *  Mariana Hernández
 *  Alan Córdova
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define STRIDE       32
#define OFFSET        0
#define GROUP_SIZE  512
#define SHARED_SIZE 256

// tamanio
#define n 8
//#define m 8

/* Utilidad para checar errores de CUDA */
void checkCUDAError(const char*);

__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N, int sec_size) {

    __shared__ float AS[SHARED_SIZE];
    __shared__ float BS[SHARED_SIZE];

    int ROW = blockIdx.y*sec_size+threadIdx.y;
    int COL = blockIdx.x*sec_size+threadIdx.x;
    float tmpSum = 0;

    for(int j = 0; j < N/sec_size; j++){
        AS[threadIdx.y * sec_size + threadIdx.x] = A[ROW * N + (j * sec_size + threadIdx.x)];
        BS[threadIdx.y * sec_size + threadIdx.x] = B[COL + (j * sec_size * N) + (threadIdx.y*N)];
        __syncthreads();

        for (int i = 0; i < sec_size; i++) {
            tmpSum += AS[threadIdx.y * sec_size + i] * BS[i * sec_size + threadIdx.x];
        }
        __syncthreads();
    }
    C[ROW * N + COL] = tmpSum;
}

// main routine that executes on the host
int main(void)
{
	float *a_h, *a_d;  // Pointer to host & device arrays
	float *mat1_h, *mat2_h, *mat_res_h, *mat1_d, *mat2_d, *mat_res_d;



	const int N = 1<<10;  // Make a big array with 2**N elements
	size_t size = N * sizeof(float);

	const int n_mat = n * n;
    size_t sz = n_mat * sizeof(float);
    /* Auxiliares para medir tiempos */

    hipEvent_t start, stop;
    float time;

    a_h = (float *)malloc(size);        // Allocate array on host
	hipMalloc((void **) &a_d, size);   // Allocate array on device

    mat1_h = (float *)malloc(sz);        // Allocate array on host
    mat2_h = (float *)malloc(sz);        // Allocate array on host
    mat_res_h = (float *)malloc(sz);        // Allocate array on host

	hipMalloc((void **) &mat1_d, sz);   // Allocate array on device
	hipMalloc((void **) &mat2_d, sz);   // Allocate array on device
	hipMalloc((void **) &mat_res_d, sz);   // Allocate array on device


    // Initialize host array and copy it to CUDA device
	for (int i=0; i<N; i++){

        a_h[i] = (float)i;

    }
    for (int i = 0; i < n_mat; ++i){

    	mat1_h[i] = i % 8;
    	mat2_h[i] = i % 8;
    	mat_res_h[i] = 0;
    }


    printf("mats:\n");
    for (int i = 0; i < n_mat; ++i){

    	if(i%n == 0)
    		printf("\n");
    	printf("%.2f ", mat1_h[i] );
    }

	hipMemcpy(mat1_d, mat1_h, sz, hipMemcpyHostToDevice);
	hipMemcpy(mat2_d, mat2_h, sz, hipMemcpyHostToDevice);
	hipMemcpy(mat_res_d, mat_res_h, sz, hipMemcpyHostToDevice);
    checkCUDAError("memcpy");

	// Create timer for timing CUDA calculation
	//PPunsigned int timer = 0;
	//PPcutCreateTimer( &timer );
	hipEventCreate(&start);
	hipEventCreate(&stop);

    // Set number of threads and blocks
	//int n_threads_per_block = 128;//1<<9;  // 512 threads per block
	//int n_blocks = 256;//1<<10;  // 1024 blocks

	// Do calculation on device

	hipEventRecord(start,0);
	//matrix_mult <<< n_blocks, n_threads_per_block >>> (mat1_d, mat2_d, mat_res_d, n);

	dim3 threadsPerBlock(n, n);
    dim3 blocksPerGrid(1, 1);

    if (n*n > 512){
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(n)/double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(n)/double(threadsPerBlock.y));
    }

    matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(mat1_d, mat2_d, mat_res_d, n, threadsPerBlock.x);


	hipDeviceSynchronize();  // Wait for matrix_mult to finish on CUDA

    checkCUDAError("kernel invocation");


	// Retrieve result from device and store it in host array
	hipMemcpy(mat1_h, mat1_d, sz, hipMemcpyDeviceToHost);
	hipMemcpy(mat2_h, mat2_d, sz, hipMemcpyDeviceToHost);
	hipMemcpy(mat_res_h, mat_res_d, sz, hipMemcpyDeviceToHost);
    checkCUDAError("memcpy");

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime( &time, start, stop );

	// Print some of the results
	//for (int i=0; i<N; i+=N/50) printf("%d %f\n", i, a_h[i]);

    // Imprime tiempo de ejecución
    printf("\n\nTIEMPO DE EJECUCIÓN: %f mSeg\n\n", time);

	printf("res:\n");
    for (int i = 0; i < n_mat; ++i)
    {
    	if(i%n == 0)
    		printf("\n");
    	printf("%.2f ", mat_res_h[i] );
    }

    hipEventDestroy( start );
    hipEventDestroy( stop );

	free(mat1_h);
	free(mat2_h);
	free(mat_res_h);

	hipFree(mat1_d);
	hipFree(mat2_d);
	hipFree(mat_res_d);
}

/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
